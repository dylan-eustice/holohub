#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "adv_networking_tx.h"  //todo: Rename networking connectors

namespace holoscan::ops {

void AdvConnectorOpTx::setup(OperatorSpec& spec) {
  spec.input<std::shared_ptr<RFChannel>>("rf_in");
  spec.output<std::shared_ptr<AdvNetBurstParams>>("burst_out");

  // Packet size / type parameters
  spec.param<uint32_t>(batch_size_,
    "batch_size",
    "Batch size",
    "Batch size for each processing epoch", 1000);
  spec.param<uint16_t>(payload_size_,
    "payload_size",
    "Payload size",
    "Payload size to send. Does not include <= L4 headers", 1400);
  spec.param<int>(hds_,
    "split_boundary",
    "Header-data split boundary",
    "Byte boundary where header and data is split", 0);
  spec.param<uint16_t>(header_size_,
    "header_size",
    "Header size",
    "Header size on each packet from L4 and below", 42);
  spec.param<bool>(gpu_direct_,
    "gpu_direct",
    "GPUDirect enabled",
    "Byte boundary where header and data is split", false);

  // Radar parameters
  spec.param(num_pulses_,
    "numPulses",
    "Number of pulses",
    "Number of pulses per channel", {});
  spec.param(num_channels_,
    "numChannels",
    "Number of channels",
    "Number of channels", {});
  spec.param(waveform_length_,
    "waveformLength",
    "Waveform length",
    "Length of waveform", {});
  spec.param(num_samples_,
    "numSamples",
    "Number of samples",
    "Number of samples per channel", {});

  // Networking parameters
  spec.param<uint16_t>(udp_src_port_,
    "udp_src_port",
    "UDP source port",
    "UDP source port");
  spec.param<uint16_t>(udp_dst_port_,
    "udp_dst_port",
    "UDP destination port",
    "UDP destination port");
  spec.param<std::string>(ip_src_addr_,
    "ip_src_addr",
    "IP source address",
    "IP source address");
  spec.param<std::string>(ip_dst_addr_,
    "ip_dst_addr",
    "IP destination address",
    "IP destination address");
  spec.param<std::string>(eth_dst_addr_,
    "eth_dst_addr",
    "Ethernet destination address",
    "Ethernet destination address");
  spec.param<uint16_t>(port_id_,
    "port_id",
    "Interface number",
    "Interface number");
}

void AdvConnectorOpTx::initialize() {
  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::initialize()");
  holoscan::Operator::initialize();

  // Compute how many packets sent per array
  samples_per_pkt = (payload_size_.get() - RFPacket::header_size()) / sizeof(complex_t);
  num_packets_buf = packets_per_channel(payload_size_.get(), num_pulses_.get(), num_samples_.get());
  HOLOSCAN_LOG_INFO("samples_per_pkt: {}", samples_per_pkt);
  HOLOSCAN_LOG_INFO("num_packets_buf: {}", num_packets_buf);

  if (num_packets_buf >= batch_size_.get()) {
    //todo: Figure out a better way to break up and send a large chunk of data
    HOLOSCAN_LOG_ERROR(
      "RF array size too large: [{}, {}] requires {} packets and the max batch size is set to {}",
      num_pulses_.get(), num_samples_.get(), num_packets_buf, batch_size_.get());
    return;
  }

  // Reserve memory
  buf_stride = RFPacket::packet_size(samples_per_pkt);
  buf_size   = num_packets_buf * buf_stride;
  if (!gpu_direct_.get()) {
    // On CPU
    mem_buf_h_ = static_cast<uint8_t *>(malloc(buf_size));
    if (mem_buf_h_ == nullptr) {
      HOLOSCAN_LOG_ERROR("Failed to allocate {} bytes of CPU batch memory", buf_size);
      return;
    }
    packets_buf = new RFPacket[num_packets_buf];
    for (size_t i = 0; i < num_packets_buf; i++) {
      packets_buf[i] = RFPacket(&mem_buf_h_[i * buf_stride]);
    }
  }
  else {
    //todo: GPU-only mode
    // // On GPU
    // for (int n = 0; n < num_concurrent; n++) {
    //   hipHostMalloc(&gpu_bufs[n], sizeof(uint8_t**) * batch_size_.get());
    //   hipStreamCreate(&streams_[n]);
    //   hipEventCreate(&events_[n]);
    // }
    // HOLOSCAN_LOG_INFO("Initialized {} streams and events", num_concurrent);
  }

  adv_net_format_eth_addr(eth_dst_, eth_dst_addr_.get());
  inet_pton(AF_INET, ip_src_addr_.get().c_str(), &ip_src_);
  inet_pton(AF_INET, ip_dst_addr_.get().c_str(), &ip_dst_);

  // ANO expects host order when setting
  ip_src_ = ntohl(ip_src_);
  ip_dst_ = ntohl(ip_dst_);

  //todo: GPU-only mode Eth+IP+UDP headers

  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::initialize() complete");
}

AdvNetStatus AdvConnectorOpTx::set_cpu_hdr(AdvNetBurstParams *msg, const int pkt_idx) {
  AdvNetStatus ret;

  // Set Ethernet header
  if ((ret = adv_net_set_cpu_eth_hdr(msg,
                                     pkt_idx,
                                     eth_dst_)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set Ethernet header for packet {}", pkt_idx);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  // Remove Eth + IP size
  const auto ip_len = payload_size_.get() + header_size_.get() - (14 + 20);
  if ((ret = adv_net_set_cpu_ipv4_hdr(msg,
                                      pkt_idx,
                                      ip_len,
                                      17,
                                      ip_src_,
                                      ip_dst_)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set IP header for packet {}", 0);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  // Set UDP header
  if ((ret = adv_net_set_cpu_udp_hdr(msg,
                                     pkt_idx,
                                     // Remove Eth + IP + UDP headers
                                     payload_size_.get() + header_size_.get() - (14 + 20 + 8),
                                     udp_src_port_.get(),
                                     udp_dst_port_.get())) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Failed to set UDP header for packet {}", 0);
    adv_net_free_all_burst_pkts_and_burst(msg);
    return ret;
  }

  return AdvNetStatus::SUCCESS;
}

void AdvConnectorOpTx::compute(InputContext& op_input,
                               OutputContext& op_output,
                               ExecutionContext& context) {
  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::compute()");
  AdvNetStatus ret;

  // Check if GPU send is falling behind
  if (gpu_direct_.get() && (hipEventQuery(events_[cur_idx]) != hipSuccess)) {
    //todo: GPU-only mode
    // HOLOSCAN_LOG_ERROR("Falling behind on TX processing for index {}!", cur_idx);
    // return;
  }

  // Input is pulse/sample data from a single channel
  auto rf_data = op_input.receive<std::shared_ptr<RFChannel>>("rf_in").value();

  /**
   * Spin waiting until a buffer is free. This can be stalled by sending
   * faster than the NIC can handle it. We expect the transmit operator to
   * operate much faster than the receiver since it's not having to do any
   * work to construct packets, and just copying from a buffer into memory.
  */
  auto msg = adv_net_create_burst_params();
  adv_net_set_hdr(msg, port_id_.get(), queue_id, num_packets_buf);

  while (!adv_net_tx_burst_available(msg)) {}
  if ((ret = adv_net_get_tx_pkt_burst(msg)) != AdvNetStatus::SUCCESS) {
    HOLOSCAN_LOG_ERROR("Error returned from adv_net_get_tx_pkt_burst: {}",
      static_cast<int>(ret));
    return;
  }

  // Generate packets from RF data //todo Optimize this process
  index_t ix_buf = 0;
  index_t ix_max = static_cast<index_t>(num_samples_.get());
  for (index_t ix_pulse = 0; ix_pulse < num_pulses_.get(); ix_pulse++) {
    for (index_t ix_sample = 0; ix_sample < num_samples_.get(); ix_sample += samples_per_pkt) {
      // Slice to the samples this packet will send
      auto data = rf_data->data.Slice<1>(
        {ix_pulse, ix_sample},
        {matxDropDim, std::min(ix_sample + samples_per_pkt, ix_max)});

      // Use accessor functions to set payload
      packets_buf[ix_buf].set_waveform_id(rf_data->waveform_id);
      packets_buf[ix_buf].set_sample_idx(ix_sample);
      packets_buf[ix_buf].set_channel_idx(rf_data->channel_id);
      packets_buf[ix_buf].set_pulse_idx(ix_pulse);
      packets_buf[ix_buf].set_num_samples(data.Size(0));
      packets_buf[ix_buf].set_end_array(0);
      packets_buf[ix_buf].set_payload(data.Data(), rf_data->stream);
      ix_buf++;
    }
  }
  if (num_packets_buf != ix_buf || adv_net_get_num_pkts(msg) != ix_buf) {
    HOLOSCAN_LOG_ERROR("Not sending expected number of packets");
  }

  // Send end-of-array message if this is the last channel of the transmit
  const bool send_eoa_msg = rf_data->channel_id == (num_channels_.get() - 1);
  if (send_eoa_msg) {
    packets_buf[num_packets_buf - 1].set_end_array(1);
  }

  // Setup packets
  int cpu_len;
  int gpu_len;
  for (int pkt_idx = 0; pkt_idx < adv_net_get_num_pkts(msg); pkt_idx++) {
    // For HDS mode or CPU mode populate the packet headers
    if (!gpu_direct_.get() || hds_.get() > 0) {
      ret = set_cpu_hdr(msg, pkt_idx); // set packet headers
      if (ret != AdvNetStatus::SUCCESS) {
        return;
      }

      // Only set payload on CPU buffer if we're not using GPUDirect
      if (!gpu_direct_.get()) {
        if ((ret = adv_net_set_cpu_udp_payload(msg,
                                               pkt_idx,
                                               packets_buf[pkt_idx].get_ptr(),
                                               payload_size_.get())) != AdvNetStatus::SUCCESS) {
          HOLOSCAN_LOG_ERROR("Failed to set UDP payload for packet {}", pkt_idx);
          adv_net_free_all_burst_pkts_and_burst(msg);
          return;
        }
      }
    }

    // Figure out the CPU and GPU length portions for ANO
    if (gpu_direct_.get() && hds_.get() > 0) {
      //todo: GPU-only mode
      // cpu_len = hds_.get();
      // gpu_len = payload_size_.get();
      // gpu_bufs[cur_idx][pkt_idx] = reinterpret_cast<uint8_t *>(
      //   adv_net_get_gpu_pkt_ptr(msg, pkt_idx));
    }
    else if (!gpu_direct_.get()) {
      cpu_len = payload_size_.get() + header_size_.get();  // sizeof UDP header
      gpu_len = 0;
    }
    else {
      //todo: GPU-only mode
      // cpu_len = 0;
      // gpu_len = payload_size_.get() + header_size_.get();  // sizeof UDP header
      // gpu_bufs[cur_idx][pkt_idx] = reinterpret_cast<uint8_t *>(
      //   adv_net_get_gpu_pkt_ptr(msg, pkt_idx));
    }

    if ((ret = adv_net_set_pkt_len(msg, pkt_idx, cpu_len, gpu_len)) != AdvNetStatus::SUCCESS) {
      HOLOSCAN_LOG_ERROR("Failed to set lengths for packet {}", pkt_idx);
      adv_net_free_all_burst_pkts_and_burst(msg);
      return;
    }
  }

  // In GPU-only mode copy the header
  if (gpu_direct_.get() && hds_.get() == 0) {
    //todo: GPU-only mode
    // copy_headers(gpu_bufs[cur_idx], gds_header_,
    //   header_size_.get(), adv_net_get_num_pkts(msg), streams_[cur_idx]);
  }

  // Populate packets with 16-bit numbers of {0,0}, {1,1}, ...
  if (gpu_direct_.get()) {
    //todo: GPU-only mode
    // const auto offset = (hds_.get() > 0) ? 0 : header_size_.get();
    // populate_packets(gpu_bufs[cur_idx], payload_size_.get(),
    //     adv_net_get_num_pkts(msg), offset, streams_[cur_idx]);
    // hipEventRecord(events_[cur_idx], streams_[cur_idx]);
    // out_q.push(TxMsg{msg, events_[cur_idx]});
  }

  // Transmit
  HOLOSCAN_LOG_INFO("AdvConnectorOpTx sending {} packets... ({}, {})",
    adv_net_get_num_pkts(msg),
    rf_data->waveform_id,
    rf_data->channel_id);
  if (gpu_direct_.get()) {
    //todo: GPU-only mode
    // const auto first = out_q.front();
    // if (hipEventQuery(first.evt) == hipSuccess) {
    //   op_output.emit(first.msg, "burst_out");
    //   out_q.pop();
    // }
  }
  else {
    op_output.emit(msg, "burst_out");
  }

  // Increment index
  cur_idx = (++cur_idx % num_concurrent);

  HOLOSCAN_LOG_INFO("AdvConnectorOpTx::compute() done");
}

}  // namespace holoscan::ops
